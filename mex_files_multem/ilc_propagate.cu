#include "hip/hip_runtime.h"
/**
 * This file is part of MULTEM.
 * Copyright 2020 Ivan Lobato <Ivanlh20@gmail.com>
 *
 * MULTEM is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * MULTEM is distributed in the hope that it will be useful, 
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with MULTEM. If not, see <http:// www.gnu.org/licenses/>.
 */

#include "types.cuh"
#include "matlab_types.cuh"
#include "traits.cuh"
#include "stream.cuh"
#include "fft.cuh"
#include "input_multislice.cuh"
#include "output_multislice.hpp"

#include "propagator.cuh"

#include <mex.h>
#include "matlab_mex.cuh"

using mt::rmatrix_r;
using mt::rmatrix_c;

template <class TInput_Multislice>
void read_input_multislice(const mxArray *mx_input_multislice, TInput_Multislice &input_multislice, bool full = true)
{
	using T_r = mt::Value_type<TInput_Multislice>;

	/************************ simulation type **************************/
	input_multislice.simulation_type = mt::eTEMST_PropRS;

	/************** Electron-Phonon interaction model ******************/
	input_multislice.pn_model = mt::ePM_Still_Atom; 

	/**************************** Specimen *****************************/
	auto lx = mx_get_scalar_field<T_r>(mx_input_multislice, "spec_lx");
	auto ly = mx_get_scalar_field<T_r>(mx_input_multislice, "spec_ly");
	T_r lz = 0;
	T_r dz = 0.25;
	bool pbc_xy = true; 

	/************************** xy sampling ****************************/
	auto nx = mx_get_scalar_field<int>(mx_input_multislice, "nx");
	auto ny = mx_get_scalar_field<int>(mx_input_multislice, "ny");
	bool bwl = false;

	input_multislice.grid_2d.set_input_data(nx, ny, lx, ly, dz, bwl, pbc_xy);

	/************************ Incident wave ****************************/
	auto iw_type = mx_get_scalar_field<mt::eIncident_Wave_Type>(mx_input_multislice, "iw_type");
	input_multislice.set_incident_wave_type(iw_type);

	if(input_multislice.is_user_define_wave() && full)
	{
		auto iw_psi = mx_get_matrix_field<rmatrix_c>(mx_input_multislice, "iw_psi");
		mt::assign(iw_psi, input_multislice.iw_psi);
	}

	// read iw_x and iw_y
	auto iw_x = mx_get_matrix_field<rmatrix_r>(mx_input_multislice, "iw_x");
	auto iw_y = mx_get_matrix_field<rmatrix_r>(mx_input_multislice, "iw_y");
	
	int n_iw_xy = min(iw_x.size(), iw_y.size()); 
	input_multislice.iw_x.assign(iw_x.begin(), iw_x.begin()+n_iw_xy);
	input_multislice.iw_y.assign(iw_y.begin(), iw_y.begin()+n_iw_xy);

	/********************* Microscope parameter ***********************/
	input_multislice.E_0 = mx_get_scalar_field<T_r>(mx_input_multislice, "E_0");
	input_multislice.theta = mx_get_scalar_field<T_r>(mx_input_multislice, "theta")*mt::c_deg_2_rad;
	input_multislice.phi = mx_get_scalar_field<T_r>(mx_input_multislice, "phi")*mt::c_deg_2_rad;

	/************************ Objective lens **************************/
	input_multislice.obj_lens.c_10 = mx_get_scalar_field<T_r>(mx_input_multislice, "obj_lens_c_10"); 	// defocus(Angstrom)
	input_multislice.obj_lens.set_input_data(input_multislice.E_0, input_multislice.grid_2d);
	
	/********************* select output region *************************/
	input_multislice.output_area.ix_0 = mx_get_scalar_field<int>(mx_input_multislice, "output_area_ix_0")-1;
	input_multislice.output_area.iy_0 = mx_get_scalar_field<int>(mx_input_multislice, "output_area_iy_0")-1;
	input_multislice.output_area.ix_e = mx_get_scalar_field<int>(mx_input_multislice, "output_area_ix_e")-1;
	input_multislice.output_area.iy_e = mx_get_scalar_field<int>(mx_input_multislice, "output_area_iy_e")-1;

	/********************* validate parameters *************************/
	input_multislice.validate_parameters();
 }

template<class TOutput_Multislice>
void set_struct_propagate(TOutput_Multislice &output_multislice, mxArray *&mx_output_multislice)
{
	const char *field_names_output_multislice[] = {"dx", "dy", "x", "y", "thick", "psi"};
	int number_of_fields_output_multislice = 6;
	mwSize dims_output_multislice[2] = {1, 1};

	mx_output_multislice = mxCreateStructArray(2, dims_output_multislice, number_of_fields_output_multislice, field_names_output_multislice);

	mx_create_set_scalar_field<rmatrix_r>(mx_output_multislice, 0, "dx", output_multislice.dx);
	mx_create_set_scalar_field<rmatrix_r>(mx_output_multislice, 0, "dy", output_multislice.dy);
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "x", 1, output_multislice.x.size(), output_multislice.x);
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "y", 1, output_multislice.y.size(), output_multislice.y);
	mx_create_set_matrix_field<rmatrix_r>(mx_output_multislice, "thick", 1, output_multislice.thick.size(), output_multislice.thick);
	mx_create_set_matrix_field<rmatrix_c>(mx_output_multislice, "psi", output_multislice.ny, output_multislice.nx, output_multislice.psi_coh[0]);
}

template <class T, mt::eDevice dev>
void run_propagate(mt::System_Configuration &system_conf, const mxArray *mx_input_multislice, mxArray *&mx_output_multislice)
{
	mt::Input_Multislice<T> input_multislice;
	read_input_multislice(mx_input_multislice, input_multislice);
	input_multislice.system_conf = system_conf;

	mt::Stream<dev> stream(system_conf.nstream);
	mt::FFT<T, dev> fft_2d;
	fft_2d.create_plan_2d(input_multislice.grid_2d.ny, input_multislice.grid_2d.nx, system_conf.nstream);

	mt::Propagator<T, dev> propagator;
	propagator.set_input_data(&input_multislice, &stream, &fft_2d);

	mt::Output_Multislice<T> output_multislice;
	output_multislice.set_input_data(&input_multislice);

	propagator(mt::eS_Real, input_multislice.gx_0(), input_multislice.gy_0(), input_multislice.obj_lens.c_10, output_multislice);

	stream.synchronize();

	//output_multislice.gather();
	output_multislice.clean_temporal();
	fft_2d.cleanup();

	set_struct_propagate(output_multislice, mx_output_multislice);
}

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[])
{	
	auto system_conf = mt::read_system_conf(prhs[0]);
	int idx_0 = (system_conf.active)?1:0;

	if(system_conf.is_float_host())
	{
		run_propagate<float, mt::e_host>(system_conf, prhs[idx_0], plhs[0]);
	}
	else if(system_conf.is_double_host())
	{
		run_propagate<double, mt::e_host>(system_conf, prhs[idx_0], plhs[0]);
	}
	else if(system_conf.is_float_device())
	{
		run_propagate<float, mt::e_device>(system_conf, prhs[idx_0], plhs[0]);
	}
	else if(system_conf.is_double_device())
	{
		run_propagate<double, mt::e_device>(system_conf, prhs[idx_0], plhs[0]);
	}
}